#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>

#define CHECK_CUDA(call)                                                        \
    do {                                                                        \
        hipError_t err = call;                                                 \
        if (err != hipSuccess) {                                               \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;\
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    } while (0)

void loadCheckpointFromDisk(const std::string& filename, std::vector<float>& weights) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Failed to open checkpoint file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    file.seekg(0, std::ios::end);
    size_t fileSize = file.tellg();
    file.seekg(0, std::ios::beg);

    weights.resize(fileSize / sizeof(float));
    file.read(reinterpret_cast<char*>(weights.data()), fileSize);

    if (!file) {
        std::cerr << "Failed to read checkpoint file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    std::cout << "Loaded checkpoint with " << weights.size() << " weights." << std::endl;
}

int main() {
    // Model checkpoint filename
    const std::string checkpointFile = "/work/hdd/bdof/nkanamarla/models/LLAMA3download/model_checkpoint.bin";

    // Load model weights from disk
    std::vector<float> weights;
    loadCheckpointFromDisk(checkpointFile, weights);

    // Set the GPUs to use
    int deviceCount;
    CHECK_CUDA(hipGetDeviceCount(&deviceCount));
    if (deviceCount < 2) {
        std::cerr << "This program requires at least two GPUs." << std::endl;
        return EXIT_FAILURE;
    }

    int srcDevice = 0;
    int dstDevice = 1;

    // Allocate memory on source GPU
    CHECK_CUDA(hipSetDevice(srcDevice));
    float* d_srcWeights;
    size_t dataSize = weights.size() * sizeof(float);
    CHECK_CUDA(hipMalloc(&d_srcWeights, dataSize));
    CHECK_CUDA(hipMemcpy(d_srcWeights, weights.data(), dataSize, hipMemcpyHostToDevice));

    std::cout << "Weights transferred to GPU " << srcDevice << "." << std::endl;

    // Enable peer access between GPUs
    int canAccessPeer;
    CHECK_CUDA(hipDeviceCanAccessPeer(&canAccessPeer, dstDevice, srcDevice));
    if (canAccessPeer) {
        CHECK_CUDA(hipDeviceEnablePeerAccess(dstDevice, 0));
    } else {
        std::cerr << "Peer access not supported between GPU " << srcDevice << " and GPU " << dstDevice << "." << std::endl;
        return EXIT_FAILURE;
    }

    // Allocate memory on destination GPU
    CHECK_CUDA(hipSetDevice(dstDevice));
    float* d_dstWeights;
    CHECK_CUDA(hipMalloc(&d_dstWeights, dataSize));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Start timing
    CHECK_CUDA(hipEventRecord(start, 0));

    // Transfer data between GPUs
    CHECK_CUDA(hipMemcpyPeer(d_dstWeights, dstDevice, d_srcWeights, srcDevice, dataSize));

    // Stop timing
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "GPU-to-GPU transfer took " << milliseconds << " ms." << std::endl;

    // Cleanup
    CHECK_CUDA(hipFree(d_srcWeights));
    CHECK_CUDA(hipFree(d_dstWeights));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipSetDevice(srcDevice));
    CHECK_CUDA(hipDeviceDisablePeerAccess(dstDevice));

    std::cout << "Cleanup complete. Exiting program." << std::endl;
    return 0;
}
