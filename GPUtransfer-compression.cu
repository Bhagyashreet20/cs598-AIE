#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <nvcomp.hpp>
#include <nvcomp/cascaded.hpp>

#define CHECK_CUDA(call)                                                        \
    do {                                                                        \
        hipError_t err = call;                                                 \
        if (err != hipSuccess) {                                               \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    } while (0)

void memoryCheck(int GPUnum) {
    size_t free_memory, total_memory;
    hipError_t error = hipMemGetInfo(&free_memory, &total_memory);
    
    if (error != hipSuccess) {
        std::cerr << "hipMemGetInfo failed: " << hipGetErrorString(error) << std::endl;
    }
    
    std::cout << "For GPU " << GPUnum << " Free GPU memory: " << free_memory / (1024 * 1024) << " MB" << std::endl;
    std::cout << "For GPU " << GPUnum << " Total GPU memory: " << total_memory / (1024 * 1024) << " MB" << std::endl;
    std::cout << "For GPU " << GPUnum << " Used GPU memory: " << (total_memory - free_memory) / (1024 * 1024) << " MB" << std::endl;
}

void loadCheckpointFromDisk(const std::string& filename, std::vector<uint8_t>& data) {
    std::ifstream file(filename, std::ios::binary);
    if (!file) {
        std::cerr << "Failed to open checkpoint file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    file.seekg(0, std::ios::end);
    size_t fileSize = file.tellg();
    file.seekg(0, std::ios::beg);

    data.resize(fileSize);
    file.read(reinterpret_cast<char*>(data.data()), fileSize);

    if (!file) {
        std::cerr << "Failed to read checkpoint file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    std::cout << "Loaded checkpoint of size " << data.size() << " bytes in CPU memory." << std::endl;
}

int main() {
    // Load checkpoint from disk and split for test 
    const std::string checkpointFile = "/work/hdd/bdof/nkanamarla/models/LLAMA3checkpointbinformat/LLAMA3checkpoint.bin";
    std::vector<uint8_t> weights;
    loadCheckpointFromDisk(checkpointFile, weights);

    // Split up checkpoint code for next part
    auto middle = weights.begin() + weights.size() / 2;
    std::vector<uint8_t> weightsFirstHalf(weights.begin(), middle);
    std::vector<uint8_t> weightsSecondHalf(middle, weights.end());

    int deviceCount;
    CHECK_CUDA(hipGetDeviceCount(&deviceCount));
    if (deviceCount < 2) {
        std::cerr << "This program requires at least two GPUs." << std::endl;
        return EXIT_FAILURE;
    }

    int srcDevice = 0;
    int dstDevice = 1;

    // Enable peer access between GPUs
    int canAccessPeer;
    CHECK_CUDA(hipDeviceCanAccessPeer(&canAccessPeer, srcDevice, dstDevice));
    if (canAccessPeer) {
        CHECK_CUDA(hipSetDevice(srcDevice));
        CHECK_CUDA(hipDeviceEnablePeerAccess(dstDevice, 0));
    } else {
        std::cerr << "Peer access not supported between GPU " << srcDevice << " and GPU " << dstDevice << "." << std::endl;
        return EXIT_FAILURE;
    }
    std::cout << "Setup peer access between source and destination GPU." << std::endl;
    memoryCheck(srcDevice);

    // Set up CUDA stream
    CHECK_CUDA(hipSetDevice(srcDevice));
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    // Allocate and copy data to source GPU
    uint8_t* d_srcWeights;
    size_t dataSize = weightsFirstHalf.size();
    CHECK_CUDA(hipMalloc(&d_srcWeights, dataSize));
    CHECK_CUDA(hipMemcpy(d_srcWeights, weightsFirstHalf.data(), dataSize, hipMemcpyHostToDevice));
    std::cout << "Shard weights transferred to source GPU " << srcDevice << " of size " << dataSize << " bytes." << std::endl;
    memoryCheck(srcDevice);

    {
    // Start timing
    CHECK_CUDA(hipEventRecord(start, stream));

    // Set up compression manager
    nvcompBatchedCascadedOpts_t cascade_options;
    cascade_options.type =  nvcomp::TypeOf<uint8_t>();
    cascade_options.num_RLEs = 1;
    cascade_options.num_deltas = 1;
    cascade_options.use_bp = 1;
    const size_t chunk_size = 1 << 22; // 4 MB chunks
    nvcomp::CascadedManager cascade_manager(
        chunk_size,
        cascade_options,
        stream
    );

    // Compress data on source GPU
    nvcomp::CompressionConfig comp_config = cascade_manager.configure_compression(dataSize);
    uint8_t* d_compressedData;
    comp_config.max_compressed_buffer_size = dataSize * 0.1; // nvCOMP is too conservative with the predicted compression size 
    std::cout << "Compression max possible size in bytes " << comp_config.max_compressed_buffer_size << std::endl;
    CHECK_CUDA(hipMallocAsync(&d_compressedData, comp_config.max_compressed_buffer_size, stream));
    CHECK_CUDA(hipStreamSynchronize(stream));
    try {
        cascade_manager.compress(
            d_srcWeights,
            d_compressedData,
            comp_config
    );
    } catch (const std::exception& e) {
        std::cerr << "Compression failed: " << e.what() << std::endl;
        exit(EXIT_FAILURE);
    }
    CHECK_CUDA(hipStreamSynchronize(stream));
    size_t compressed_size = cascade_manager.get_compressed_output_size(d_compressedData);
    memoryCheck(srcDevice);

    // Allocate memory on destination GPU
    CHECK_CUDA(hipSetDevice(dstDevice));
    uint8_t* d_dstCompressedData;
    CHECK_CUDA(hipMalloc(&d_dstCompressedData, compressed_size));
    memoryCheck(dstDevice);

    // Transfer compressed data between GPUs
    CHECK_CUDA(hipSetDevice(srcDevice));
    CHECK_CUDA(hipMemcpyPeerAsync(d_dstCompressedData, dstDevice, d_compressedData, srcDevice, compressed_size, stream));

    // Stop timing
    CHECK_CUDA(hipEventRecord(stop, stream));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Compressed data on GPU of size " << compressed_size << " bytes for a compression ratio of " << dataSize/compressed_size << " X." << std::endl;
    std::cout << "Data Compression and GPU-to-GPU data transfer took " << milliseconds << " ms." << std::endl;
    CHECK_CUDA(hipFree(d_compressedData));
    CHECK_CUDA(hipSetDevice(dstDevice));
    CHECK_CUDA(hipFree(d_dstCompressedData));
    } // cascade_manager is destroyed here

    // Cleanup
    CHECK_CUDA(hipFree(d_srcWeights));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipStreamDestroy(stream));
    CHECK_CUDA(hipSetDevice(srcDevice));
    CHECK_CUDA(hipDeviceDisablePeerAccess(dstDevice));

    std::cout << "Cleanup complete. Exiting program." << std::endl;
    return 0;
}